#include "hip/hip_runtime.h"
/* CUDA blur
 * Kevin Yuh, 2014 */

#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_convolve.cuh"


/* 
Atomic-max function. You may find it useful for normalization.

We haven't really talked about this yet, but __device__ functions not
only are run on the GPU, but are called from within a kernel.

Source: 
http://stackoverflow.com/questions/17399119/
cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
*/
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}



__global__
void
cudaProdScaleKernel(const hipfftComplex *raw_data, const hipfftComplex *impulse_v, 
    hipfftComplex *out_data,
    const unsigned int padded_length,
    const unsigned int impulse_len) 
{
    
    int numThreads = gridDim.x*blockDim.x;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    while(idx < padded_length)
    {
	    float sumReal = 0.0;
	    float sumImg = 0.0;
	    int impulseIdx = 0;
    	for(int j = idx; j >= 0 && impulseIdx < impulse_len; --j)
	    {
            sumReal += (raw_data[j].x*impulse_v[impulseIdx].x-raw_data[j].y*impulse_v[impulseIdx].y);
            sumImg += (raw_data[j].x*impulse_v[impulseIdx].y+raw_data[j].y*impulse_v[impulseIdx].x);
	        ++impulseIdx; 
        }
        out_data[idx].x = sumReal; 
        out_data[idx].y = sumImg;
	    idx += numThreads;
    }
}

__global__
void
cudaMaximumKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2: Implement the maximum-finding and subsequent
    normalization (dividing by maximum).

    There are many ways to do this reduction, and some methods
    have much better performance than others. 

    For this section: Please explain your approach to the reduction,
    including why you chose the optimizations you did
    (especially as they relate to GPU hardware).

    You'll likely find the above atomicMax function helpful.
    (CUDA's atomicMax function doesn't work for floating-point values.)
    It's based on two principles:
        1) From Week 2, any atomic function can be implemented using
        atomic compare-and-swap.
        2) One can "represent" floating-point values as integers in
        a way that preserves comparison, if the sign of the two
        values is the same. (see http://stackoverflow.com/questions/
        29596797/can-the-return-value-of-float-as-int-be-used-to-
        compare-float-in-cuda)

    */
    int numThreads = gridDim.x*blockDim.x;
    int sizeOfChunk = (padded_length+numThreads-1)/numThreads;
    int idx = (blockDim.x*blockIdx.x + threadIdx.x)*sizeOfChunk;
    int iter = 0;
    float local_max = out_data[0].x;
    while(idx+iter < padded_length && iter < sizeOfChunk)
    {
        if(out_data[idx+iter].x > local_max) local_max = out_data[idx+iter].x;
        ++iter;
    }  
    atomicMax(max_abs_val,local_max);
}

__global__
void
cudaDivideKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2: Implement the division kernel. Divide all
    data by the value pointed to by max_abs_val. 

    This kernel should be quite short.
    */

    int numThreads = gridDim.x*blockDim.x;
    int sizeOfChunk = (padded_length+numThreads-1)/numThreads;
    int idx = (blockDim.x*blockIdx.x + threadIdx.x)*sizeOfChunk;
    int iter = 0;
    float max_val_modified = 0.99999/(*max_abs_val);
    while(idx+iter < padded_length && iter < sizeOfChunk)
    {
        out_data[idx+iter].x *= max_val_modified;
        ++iter;
    }  
}


void cudaCallProdScaleKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const hipfftComplex *raw_data,
        const hipfftComplex *impulse_v,
        hipfftComplex *out_data,
        const unsigned int padded_length,
        const unsigned int impulse_len) {
        

    /* TODO: Call the element-wise product and scaling kernel. */
    cudaProdScaleKernel<<<blocks,threadsPerBlock>>>(raw_data, impulse_v, out_data, padded_length, impulse_len);
}

void cudaCallMaximumKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        

    /* TODO 2: Call the max-finding kernel. */
    cudaMaximumKernel<<<blocks,threadsPerBlock>>>(out_data,max_abs_val,padded_length);    
}


void cudaCallDivideKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        
    /* TODO 2: Call the division kernel. */
    cudaDivideKernel<<<blocks,threadsPerBlock>>>(out_data,max_abs_val,padded_length);
}
